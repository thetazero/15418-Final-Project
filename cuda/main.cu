#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "eval.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
__global__ void test(){
  printf("Hello Cuda World\n");
  printf("Test 1\n");
  string b1 = "x x . . .\no o . . .\n. . . . .\n. . . . .\n. . . . .";
  printf("Test 2\n");
  Engine_Board board(b1, 5, 'x');
  printf("Board:\n%s\n", board.to_string().c_str());
  int move = board.fast_engine_recommendation(3);
  printf("Move: %d\n", move);
}
*/

void print_gpu() {
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);
  printf("Number of GPUs: %d\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
    prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
    prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
    2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

int main(int argc, char** argv )
{
   //test<<<1,1>>>();
   gpuErrchk(hipPeekAtLastError());
   hipDeviceSynchronize();
   print_gpu();

   hello_world<<3,3>>();
   return 0;
}
