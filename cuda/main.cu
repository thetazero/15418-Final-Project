#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "board.h"
#include "engine_board.h"

__global__ void test(){
  printf("Hi Cuda World\n");
  string b1 = R"(
x x . . .
o o . . .
. . . . .
. . . . .
. . . . .
  )"
  Engine_board board(b1, 5, 'x')
  int move = board.fast_engine_recomendation();
  printf("Move: %d\n", move);
}

void print_gpu() {
  int nDevices = 0;
  hipGetDeviceCount(&nDevices);
  printf("Number of GPUs: %d\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
    prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
    prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
    2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

int main(int argc, char** argv )
{
   test<<<1,1>>>();
   hipDeviceSynchronize();
   print_gpu();
   return 0;
}