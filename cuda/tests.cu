#include "hip/hip_runtime.h"
#include "eval.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void idx_kernel(int *ans, int r, int c, int size) {
  *ans = idx(r, c, size);
}

int idx_wrapper(int r, int c, int size) {
  int h_ans;
  int *d_ans;
  hipMalloc(&d_ans, sizeof(int));
  idx_kernel<<<1, 1>>>(d_ans, r, c, size);
  hipMemcpy(&h_ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_ans);
  return h_ans;
}

char *to_device(char *h_arr, int size) {
  char *d_arr;
  hipMalloc(&d_arr, size);
  hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);
  return d_arr;
}

char *to_host(char *d_arr, int size) {
  char *h_arr = new char[size];
  hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
  return h_arr;
}

__global__ void scan_horizontal_kernel(int size, char *board, char *x_scratch, char *o_scratch, int dc){
  scan_horizontal(size, board, x_scratch, o_scratch, dc);
}

void scan_horizontal_wrapper(int size, char *board, char *x_scratch,
                             char *o_scratch, int dc) {
  size_t board_mem_size = size * size * sizeof(char);
  char *d_board = to_device(board, board_mem_size);
  char *d_x_scratch = to_device(x_scratch, board_mem_size);
  char *d_o_scratch = to_device(o_scratch, board_mem_size);
  scan_horizontal_kernel<<<1, 1>>>(size, d_board, d_x_scratch, d_o_scratch, dc);
  hipMemcpy(board, d_board, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(x_scratch, d_x_scratch, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(o_scratch, d_o_scratch, board_mem_size, hipMemcpyDeviceToHost);
}

void test_idx() {
  assert(idx_wrapper(1, 4, 5) == 9);
  assert(idx_wrapper(0, 0, 5) == 0);
  assert(idx_wrapper(0, 7, 12) == 7);
  assert(idx_wrapper(8, 3, 10) == 83);
}

void test_scan_horizontal() {
  char board[9] = {'x', 'x', '.',
                   '.', 'o', '.',
                   'x', '.', 'o'};
  char x_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  char o_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  scan_horizontal_wrapper(3, board, x_scratch, o_scratch, 1);
  char expected_board[9] = {'x', 'x', '.',
                            '.', 'o', '.',
                            'x', '.', 'o'};
  char expected_x_scratch[9] = {0, 0, 2,
                                0, 0, 0,
                                0, 1, 0};
  char expected_o_scratch[9] = {0, 0, 0,
                                0, 0, 1,
                                0, 0, 0};
  scan_horizontal_wrapper(3, board, x_scratch, o_scratch, 1);
  for (int i = 0; i < 9; i++) {
    assert(board[i] == expected_board[i]);
    assert(x_scratch[i] == expected_x_scratch[i]);
    assert(o_scratch[i] == expected_o_scratch[i]);
  }
}

int main() {
  test_idx();
  test_scan_horizontal();
  return 0;
}
