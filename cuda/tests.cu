#include "hip/hip_runtime.h"
#include "eval.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void idx_kernel(int *ans, int r, int c, int size) {
  *ans = idx(r, c, size);
}

int idx_wrapper(int r, int c, int size) {
  int h_ans;
  int *d_ans;
  hipMalloc(&d_ans, sizeof(int));
  idx_kernel<<<1, 1>>>(d_ans, r, c, size);
  hipMemcpy(&h_ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_ans);
  return h_ans;
}

char *to_device(char *h_arr, int size) {
  char *d_arr;
  hipMalloc(&d_arr, size);
  hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);
  return d_arr;
}

char *to_host(char *d_arr, int size) {
  char *h_arr = new char[size];
  hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);
  return h_arr;
}

__global__ void scan_horizontal_kernel(int size, char *board, char *x_scratch, char *o_scratch, int dc){
  scan_horizontal(size, board, x_scratch, o_scratch, dc);
}

void scan_horizontal_wrapper(int size, char *board, char *x_scratch,
                             char *o_scratch, int dc) {
  size_t board_mem_size = size * size * sizeof(char);
  char *d_board = to_device(board, board_mem_size);
  char *d_x_scratch = to_device(x_scratch, board_mem_size);
  char *d_o_scratch = to_device(o_scratch, board_mem_size);
  scan_horizontal_kernel<<<1, 1>>>(size, d_board, d_x_scratch, d_o_scratch, dc);
  hipMemcpy(board, d_board, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(x_scratch, d_x_scratch, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(o_scratch, d_o_scratch, board_mem_size, hipMemcpyDeviceToHost);
}

__global__
void scan_vertical_kernel(int size, char *board, char *x_scratch, char *o_scratch, int dr){
  scan_vertical(size, board, x_scratch, o_scratch, dr);
}

void scan_vertical_wrapper(int size, char *board, char *x_scratch, char *o_scratch, int dr){
  size_t board_mem_size = size * size * sizeof(char);
  char *d_board = to_device(board, board_mem_size);
  char *d_x_scratch = to_device(x_scratch, board_mem_size);
  char *d_o_scratch = to_device(o_scratch, board_mem_size);
  scan_vertical_kernel<<<1, 1>>>(size, d_board, d_x_scratch, d_o_scratch, dr);
  hipMemcpy(board, d_board, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(x_scratch, d_x_scratch, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(o_scratch, d_o_scratch, board_mem_size, hipMemcpyDeviceToHost);
}


void test_idx() {
  assert(idx_wrapper(1, 4, 5) == 9);
  assert(idx_wrapper(0, 0, 5) == 0);
  assert(idx_wrapper(0, 7, 12) == 7);
  assert(idx_wrapper(8, 3, 10) == 83);
}

void test_scan_helper(string name, int size, char *board, char *x_scratch, char *o_scratch,
                                 char *expected_board, char *expected_x_scratch,
                                 char *expected_o_scratch, int dr, int dc) {
  cout << "Running test: " << name << endl;
  if (dr == 0) {
    scan_horizontal_wrapper(size, board, x_scratch, o_scratch, dc);
  } else if (dc == 0) {
    scan_vertical_wrapper(size, board, x_scratch, o_scratch, dr);
  } else {
    assert(false);
  }
  for (int i = 0; i < size * size; i++) {
    cout << "Got: " << board[i] << (int)x_scratch[i] << (int)o_scratch[i] << endl;
    cout << "Expect: " << expected_board[i] << (int)expected_x_scratch[i] << (int)expected_o_scratch[i] << endl;
    assert(board[i] == expected_board[i]);
    assert(x_scratch[i] == expected_x_scratch[i]);
    assert(o_scratch[i] == expected_o_scratch[i]);
  }
}

void test_scan_horizontal() {
  char board[9] = {'x', 'x', '.',
                   '.', 'o', '.',
                   'x', '.', 'o'};
  char x_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  char o_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  char expected_board[9] = {'x', 'x', '.',
                            '.', 'o', '.',
                            'x', '.', 'o'};
  char expected_x_scratch[9] = {0, 0, 2,
                                0, 0, 0,
                                0, 1, 0};
  char expected_o_scratch[9] = {0, 0, 0,
                                0, 0, 1,
                                0, 0, 0};
  test_scan_helper("scan right", 3, board, x_scratch, o_scratch,
    expected_board, expected_x_scratch, expected_o_scratch, 0, 1);

  char e_xs[9] = {0, 0, 2, 0, 0, 0, 0, 1, 0};
  char e_os[9] = {0, 0, 0, 1, 0, 1, 0, 1, 0};
  // scan same board with previous scratch but from left
  test_scan_helper("scan left", 3, board, x_scratch, o_scratch,
    expected_board, e_xs, e_os, 0, -1);
}

void test_scan_vertical() {
  char board[9] = {'x', 'x', '.',
                   '.', 'o', '.',
                   'x', '.', 'o'};
  char x_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  char o_scratch[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
  char expected_board[9] = {'x', 'x', '.',
                            '.', 'o', '.',
                            'x', '.', 'o'};
  char expected_x_scratch[9] = {0, 0, 0,
                                1, 0, 0,
                                0, 0, 0};
  char expected_o_scratch[9] = {0, 0, 0,
                                0, 0, 0,
                                0, 1, 0};
  test_scan_helper("scan down", 3, board, x_scratch, o_scratch,
    expected_board, expected_x_scratch, expected_o_scratch, 1, 0);

  char e_xs[9] = {0, 0, 0,
                  1, 0, 0,
                  0, 0, 0};
  char e_os[9] = {0, 0, 0,
                  0, 0, 1,
                  0, 1, 0};
  // scan same board with previous scratch but from left
  test_scan_helper("scan up", 3, board, x_scratch, o_scratch,
    expected_board, e_xs, e_os, -1, 0);

}

__global__
void scan_all_kernel(int size, char *board, char *x_scratch, char *o_scratch) {
  scan_all(size, board, x_scratch, o_scratch);
}

void scan_all_wrapper(int size, char *board, char *x_scratch, char *o_scratch){
  size_t board_mem_size = size * size * sizeof(char);
  char *d_board = to_device(board, board_mem_size);
  char *d_x_scratch = to_device(x_scratch, board_mem_size);
  char *d_o_scratch = to_device(o_scratch, board_mem_size);
  scan_all_kernel<<<1, 1>>>(size, d_board, d_x_scratch, d_o_scratch);
  hipMemcpy(board, d_board, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(x_scratch, d_x_scratch, board_mem_size, hipMemcpyDeviceToHost);
  hipMemcpy(o_scratch, d_o_scratch, board_mem_size, hipMemcpyDeviceToHost);
}

void test_scan_all_helper(string name, int size, char *board, char *x_scratch, char *o_scratch,
                                 char *expected_board, char *expected_x_scratch,
                                 char *expected_o_scratch) {
  cout << "Running test: " << name << endl;
  scan_all_wrapper(size, board, x_scratch, o_scratch);
  for (int i = 0; i < size * size; i++) {
    cout << i << endl;
    cout << "Got: " << board[i] << (int)x_scratch[i] << (int)o_scratch[i] << endl;
    cout << "Expect: " << expected_board[i] << (int)expected_x_scratch[i] << (int)expected_o_scratch[i] << endl;
    assert(board[i] == expected_board[i]);
    assert(x_scratch[i] == expected_x_scratch[i]);
    assert(o_scratch[i] == expected_o_scratch[i]);
  }
}

void test_scan_all() {
  size_t s = 16;
  char b[s] = {'x', 'x', '.', '.',
               '.', 'o', '.', 'o',
               'x', '.', 'o', 'x',
               'x', '.', 'o', '.'};
  char xs[s] = {0, 0, 0, 0,
                0, 0, 0, 0,
                0, 0, 0, 0,
                0, 0, 0, 0};
  char os[s] = {0, 0, 0, 0,
                0, 0, 0, 0,
                0, 0, 0, 0,
                0, 0, 0, 0};
  char e_b[s] = {'x', 'x', '.', '.',
                 '.', 'o', '.', 'o',
                 'x', '.', 'o', 'x',
                 'x', '.', 'o', '.'};
  char e_xs[s] = {0, 0, 2, 0,
                  2, 0, 1, 0,
                  0, 1, 0, 0,
                  0, 1, 0, 1};
  char e_os[s] = {0, 0, 1, 1,
                  1, 0, 2, 0,
                  0, 1, 0, 0,
                  0, 2, 0, 2};
  test_scan_all_helper("scan all", 4, b, xs, os,
    e_b, e_xs, e_os);
}

int main() {
  test_idx();
  test_scan_horizontal();
  test_scan_vertical();
  test_scan_all();
  return 0;
}
